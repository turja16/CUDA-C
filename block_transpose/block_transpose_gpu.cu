#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define BLOCK_WIDTH 6


__global__ void blockTransposeKernel(float* A_elements, unsigned int A_width, unsigned int A_height)
{
	__shared__ float blockA[BLOCK_WIDTH][BLOCK_WIDTH];

	if( (blockDim.x * blockIdx.x + threadIdx.x < A_width) && (blockDim.y * blockIdx.y + threadIdx.y < A_height))
	{
		int baseIdx = blockDim.x * blockIdx.x + threadIdx.x;
		baseIdx += (blockDim.y * blockIdx.y + threadIdx.y)*A_width;
		blockA[threadIdx.y][threadIdx.x]= A_elements[baseIdx];
		__syncthreads();



		A_elements[baseIdx]= blockA[threadIdx.x][threadIdx.y];
	}

}
void blockTranspose(float* h_A, unsigned int A_width, unsigned int A_height)
{
	float* d_A;
	int size = A_height*A_width * sizeof(float);
	hipMalloc((void**)&d_A, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH);
	dim3 dimGrid(A_width/dimBlock.x,A_height/dimBlock.y);
	blockTransposeKernel<<<dimGrid, dimBlock>>>(d_A, A_width, A_height);	

	hipMemcpy(h_A , d_A, size, hipMemcpyDeviceToHost);

	hipFree(d_A);

}



int main()
{
	int A_width = 2*BLOCK_WIDTH;
	int A_height = 3*BLOCK_WIDTH;
	size_t size = A_height*A_width * sizeof(float);

	float* h_A = (float*)malloc(size);
	printf("Input Matrix A:\n");
	for(int row = 0; row <A_height; row++){
		for(int col=0; col < A_width; col++)
		{
			int offset = row*A_width+col;
			h_A[offset] = offset;
			printf("%.1f\t",h_A[offset]);
		}
		printf("\n");
	}



	blockTranspose(h_A, A_width,A_height);
	printf("output Matrix A:\n");
	for(int row = 0; row <A_height; row++){
		for(int col =0; col < A_width; col++)
		{       
			int offset = row*A_width+col;
			printf("%.1f\t",h_A[offset]);
		}
		printf("\n");
	}


	hipFree(h_A);
	return 0;

}
